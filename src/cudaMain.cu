#include "hip/hip_runtime.h"
// Logger Lib
#define SPDLOG_ACTIVE_LEVEL SPDLOG_LEVEL_TRACE
#include <spdlog/spdlog.h>
#include <GL/gl.h>

#include "cudaMain.cuh"

// Texture ID
GLuint cudaTexID;

double timeStart;

int device = 0;

int *h_board;
int *d_board;
int *d_nextboard;
int rows;
int cols;
int size;
int threadsPerBlock;

GLuint genCudaTexImage() {
    GLuint tid;
    glGenTextures(1, &tid);
    glBindTexture(GL_TEXTURE_2D, tid);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0,  GL_RED,
                 GL_FLOAT, h_dataA);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);

    glBindTexture(GL_TEXTURE_2D, tid);
    return tid;
}

__host__ int  cpu_get_x_lined(int * board, int col,int y,int x)
{
    return   board[(y * col) + x-1]+
             board[(y * col) + x]+
             board[(y * col) + x+1];
}

__host__ int  cpu_get_y_lined(int * board, int col,int y,int x) {
    return board[(y * col + 1) + x] +
           board[(y * col) + x] +
           board[(y * col - 1) + x];
}

__host__ int cpuDeadorAlive(int value, int currentLocation){
    if (currentLocation == 1)
        if (value - 1 == 2 || value - 1 == 3)
            return 1;
        else
            return 0;
    else if (value == 3)
        return 1;
    else
        return 0;
}

// the y and x values should be the max-1 and are static for this
// function
__host__ void cornerHost(int* board, int * nextboard,int y, int x)
{
    int value=0;
    // the first four or the corners of the main box (warps)
    //left corner of front to top and left (warps)
    value = board[0]+
            board[0+1]+

            board[4*(x/6)-1]+
            board[0]+
            board[1]+

            board[(1)*x+(4*(x/6)-1)]+
            board[(1)*x+0]+
            board[(1)*x+0+1];

    nextboard[0]= cpuDeadorAlive(value,board[0]);

    // Right top corner of Left to top and front (warps)
    value = board[x*(y-1)+((x/3)*2)]+
            board[x*(y)+((x/3)*2)]+

            board[0*x+(4*(x/6)-1)-1]+
            board[0*x+(4*(x/6)-1)]+
            board[0*x+0]+

            board[(0+1)*x+(4*(x/6)-1)-1]+
            board[(0+1)*x+(4*(x/6)-1)]+
            board[(0+1)*x+0+1];

    nextboard[0*x+(4*(x/6)-1)]= cpuDeadorAlive(value,board[0*x+(4*(x/6)-1)]);

    // left bot corner of front with left and bot (warps)

    value = board[(y-1)*x+(4*(x/6)-1)]+
            board[(y-1)*x+((x/3)*2)]+
            board[(y-1)*x+((x/3)*2)+1]+

            board[y*x+(4*(x/6)-1)]+
            board[y*x+0]+
            board[y*x+0+1]+

            board[0*x+((x/6)*5)]+
            board[0*x+((x/6)*5)+1];

            nextboard[y*x+0]= cpuDeadorAlive(value,
                                             board[y*x+0]);
    // right bot corner of left with top and bot (warps)
    value = board[(y-1)*x+(4*(x/6)-1)-1]+
            board[(y-1)*x+(4*(x/6)-1)]+
            board[(y-1)*x+0]+

            board[(y)*x+(4*(x/6)-1)-1]+
            board[(y)*x+(4*(x/6)-1)]+
            board[(y)*x+0]+

            board[0*x+((x/6)*5)]+
            board[(0+1)*x+((x/6)*5)];

    nextboard[(y)*x+(4*(x/6)-1)]= cpuDeadorAlive(value,
                                     board[(y)*x+(4*(x/6)-1)]);

    // left top corners with no wraps
    // so the next set are all left top corners
    // right with front and top
    value =     board[0*x+((x/6)*5-1)]+
            board[(0+1)*x+((x/6)*5-1)]+
            cpu_get_x_lined(board,x,0,(x/6))+
            cpu_get_x_lined(board,x,1,(x/6));

    nextboard[0*x+(x/6)]= cpuDeadorAlive(value,
                                         board[0*x+(x/6)]);

    // back with right and top
    value =     board[y*x+((x/3)*2)]+
                board[y*x+((x/3)*2)+1]+

    cpu_get_x_lined(board,x,0,(2*(x/6)))+
    cpu_get_x_lined(board,x,1,(2*(x/6)));

    nextboard[0*x+(2*(x/6))]= cpuDeadorAlive(value,
                                         board[0*x+(2*(x/6))]);

    // left with back and top
    value = board[0*x+((x/3)*2)]+
            board[(0+1)*x+((x/3)*2)]+

    cpu_get_x_lined(board,x,0,(3*(x/6)))+
    cpu_get_x_lined(board,x,1,(3*(x/6)));

    nextboard[0*x+(3*(x/6))]= cpuDeadorAlive(value,
                                             board[0*x+(3*(x/6))]);

    // the next set is the right top corners
    // front with right and top
    value =     board[0*x+((x/6)*5-1)]+
            board[0*x+((x/6)*5-1)-1]+

    cpu_get_x_lined(board,x,0,(x/6-1))+
    cpu_get_x_lined(board,x,1,(x/6-1));

    nextboard[0*x+(x/6-1)]= cpuDeadorAlive(value,
                                             board[0*x+(x/6-1)]);

    // right with top and back
    value =     board[y*x+((x/6)*5-1)]+
            board[(y-1)*x+((x/6)*5-1)]+

    cpu_get_x_lined(board,x,0,(2*(x/6)-1))+
    cpu_get_x_lined(board, x,0+1,(2*(x/6)-1));

    nextboard[0*x+(2*(x/6)-1)]= cpuDeadorAlive(value,
                                           board[0*x+(2*(x/6)-1)]);


    // right corner back
    value =     board[y*x+((x/6)*5-1)]+
            board[y*x+((x/6)*5-1)-1]+

    cpu_get_x_lined(board,x,0,(3*(x/6)-1))+
    cpu_get_x_lined(board,x,0+1,(3*(x/6)-1));

    nextboard[0*x+(2*(x/6)-1)]= cpuDeadorAlive(value,
                                               board[0*x+(3*(x/6)-1)]);

    // the next set handles the left bots corners
    // right with front and bot
    value = board[0*x+(x - 1)]+
            board[0*x+(x - 1)-1]+

    cpu_get_x_lined(board,x,y,(x/6))+
    cpu_get_x_lined(board,x,y-1,(x/6));

    nextboard[y*x+(x/6-1)]= cpuDeadorAlive(value,
                                               board[y*x+(x/6-1)]);

    // back with right and bot
    value = board[(y*x) + (x - 1)]+
            board[((y-1)*x) + (x - 1)]+

    cpu_get_x_lined(board,x,y,(2*(x/6)))+
    cpu_get_x_lined(board,x,y-1,(2*(x/6)));

    nextboard[y*x+(2*(x/6)-1)]= cpuDeadorAlive(value,
                                           board[y*x+(2*(x/6)-1)]);
    // left with back and bot
    value =     board[y*x+((x/6)*5)]+
            board[(y-1)*x+((x/6)*5)+1]+

    cpu_get_x_lined(board,x,y,(3*(x/6)))+
    cpu_get_x_lined(board,x,y-1,(3*(x/6)));

    nextboard[y*x+(3*(x/6))]= cpuDeadorAlive(value,
                                               board[y*x+(3*(x/6))]);

    // bot right corners of main
    // front with right and bot

    value = board[0*x+(5*(x/6))]+
    board[(0+1)*x+(5*(x/6))]+

    cpu_get_x_lined(board,x,y,(x/6))+
    cpu_get_x_lined(board,x,y-1,(x/6));

    nextboard[y*x+(x/6)]= cpuDeadorAlive(value,
                                             board[y*x+(x/6)]);

    // right corner back
    value =     board[y*x+(x - 1)]+
            board[(y)*x+(x - 1)-1]+

            getxlinead(board, col,wrapBoxRow,backstart)+
            getxlinead(board, col,wrapBoxRow-1,backstart);
    deadorAlive(board,nextboard,col,x,wrapBoxRow,backstart);

    cpu_get_x_lined(board,x,y,(x/6))+
    cpu_get_x_lined(board,x,y-1,(x/6));

    nextboard[y*x+(x/6)]= cpuDeadorAlive(value,
                                         board[y*x+(x/6)]);
    /*
     *
    x =

    //printboard(nextboard);

    // bot left corners of main
    // left corner back
    x =     board[start*col+botend]+
            board[start*col+botend-1]+
            getxlinead(board, col,wrapBoxRow,backstart)+
            getxlinead(board, col,wrapBoxRow-1,backstart);
    deadorAlive(board,nextboard,col,x,wrapBoxRow,backstart);

    //printboard(nextboard);
    // bot left corners of main
    // left corner left
    x =     board[start*col+botstart]+
            board[(start+1)*col+botstart]+
            getxlinead(board, col,wrapBoxRow,leftstart)+
            getxlinead(board, col,wrapBoxRow-1,backstart);
    deadorAlive(board,nextboard,col,x,wrapBoxRow,leftstart);

     */

}

__device__ int get_x_lined(int * board, int col,int x,int y)
{
    return   board[(x * col) + y-1]+
             board[(x * col) + y]+
             board[(x * col) + y+1];
}

__device__ int  get_y_lined(int * board, int col,int x,int y)
{
    return   board[(x * col+1) + y]+
             board[(x * col) + y]+
             board[(x * col-1) + y];
}
__device__ int cudaAddUpLife(int * board,int col, int x, int y)
{
    return  get_x_lined(board, col, x,y+1)+
            get_x_lined(board, col, x,y)+
            get_x_lined(board, col, x,y-1);
}
__device__ int deadorAlive(int value, int currentLocation)
{
    if (currentLocation == 1)
        if (value - 1 == 2 || value - 1 == 3)
            return 1;
        else
            return 0;
    else if (value == 3)
        return 1;
    else
        return 0;
}
__device__ int addUpLife(int * board,int col, int j, int i)
{
    return  get_x_lined(board, col, j+1,i)+
            get_x_lined(board, col, j,i)+
            get_x_lined(board, col, j-1,i);
}

__global__ void two_d_conway_block(int *board, int *next_board, int cols, int x_size, int y_size)
{
    // global thread(data) row index
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // global thread(data) column index
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int index = x*cols+y;
    int total = 0;

    if(y < x_size && x < y_size)
    {
        total = addUpLife(board,cols,x,y);
        next_board[index] = deadorAlive(total,board[index]);
    }
}

__global__ void conway_edges(int *self, int *self_next, int *other, int self_idx_step, int other_idx_step, int adjacent_offset, int len) {
    //Determine the index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx>=len) return;

    //Rebase the indexes for this thread
    self = self + self_idx_step*idx;
    other = other + other_idx_step*idx;
    self_next = self_next + self_idx_step*idx;

    //Get the complete sum
    int sum =   *(other - other_idx_step) + (*other) + *(other + other_idx_step) +
                *(self  - self_idx_step ) + (*self ) + *(self  + self_idx_step ) +
                *(self + adjacent_offset - self_idx_step) + *(self + adjacent_offset) + *(self + adjacent_offset + self_idx_step);

    //Write the result
    *self_next = (sum == 3) || (sum-(*self) == 3);
}

//temporarily working on my launches here
void launchit () {
    const int edge_length = size - 2;
    const dim3 numBlocks(edge_length/blocksize + (1&&(edge_length%blocksize)), 1, 1);

    conway_edges(int *self, int *self_next, int *other, int self_idx_step, int other_idx_step, int adjacent_offset, int len);

    conway_edges(board + cols, next_board + cols, board + size*4 - 1 + cols, cols, cols, 1, edge_length); // body wrap
    conway_edges(board + size*4 - 1 + cols, next_board + size*4 - 1 + cols, board + cols, cols, cols, -1, edge_length);

    conway_edges <<<numBlocks, blocksize>>>(board + 1, next_board + 1, board + (4 * size)+1, 1, 1, cols, edge_length); //top front
    conway_edges <<<numBlocks, blocksize>>>(board + 1 + size*2, next_board + 1 + size*2, board + 5*size - 2 + cols*(rows-1), 1, -1, cols, edge_length); //top back
    conway_edges <<<numBlocks, blocksize>>>(board + 1 + size, next_board + 1 + size, board + 5*size - 1 + cols, 1, cols, cols, edge_length); //top right
    conway_edges <<<numBlocks, blocksize>>>(board + 1 + size*3, next_board + 1 + size*3, board + 4*size + cols*(rows-2), 1, (-cols), cols, edge_length); //top left

    conway_edges <<<numBlocks, blocksize>>>(board+cols*(rows-1)+1, next_board+cols*(rows-1)+1, board + size*5 + 1, 1, 1, (-cols), edge_length); // bottom front

}
// Called when setting things up before graphs loop
__host__ void cudaMainInitialize(int size) {
    SPDLOG_INFO("Initialize Cuda");
    timeStart = 0;

    //memory allocation
    cudaTexID = genCudaTexImage();
    h_board = (int *)malloc(rows*cols*sizeof(int));

    //cuda memory allocation
    hipMalloc((void**) &d_board, sizeof(int)*size);
    hipMalloc((void**) &d_nextboard, sizeof(int)*size);

    hipMemset(d_board,0,sizeof(int)*size);

    hipMemcpy(d_board,h_board, sizeof(int)*size, hipMemcpyHostToDevice);

}

// Called for every frame
__host__ void cudaMainUpdate() {
    hipError_t code = hipDeviceSynchronize();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Device Synchronize error -- {}", hipGetErrorString(code)));
    }
    // check if kernel execution generated an error
    code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }



    // copy result from device to host

    glDeleteTextures(1, &cudaTexID);
    cudaTexID = genCudaTexImage();


}

// Called as the program close
__host__ void cudaMainCleanUp() {
    SPDLOG_INFO("Clean Up Cuda");

    // check if kernel execution generated an error
    hipError_t code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }
    // copy result from device to host
    //hipMemcpy2D( h_dataA, width * sizeof(float), d_dataA, pitch, width * sizeof(float), height,hipMemcpyDeviceToHost );

    free(h_dataA);
}









//kept just in case it has usful info
__global__ void oldk1_from_hmwk( float* g_dataA, float* g_dataB, int floatpitch, int width)
{
    extern __shared__ float s_data[];
    //Write this kernel to achieve the same output as the provided k0, but you will have to use
    // shared memory.

    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    i = i + 1; //because the edge of the data is not processed

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    j = j + 1; //because the edge of the data is not processed

    int threadID = threadIdx.x;

    int s_rowwidth = blockDim.x + 2;

    // Index's

    // -- Global
    int g_ind_0 = (i-1) * floatpitch +  j;
    int g_ind_1 = i * floatpitch + j;
    int g_ind_2 = (i+1) * floatpitch +  j;

    // -- Shared
    int s_ind_0 = threadID + 1 + (s_rowwidth * 0);
    int s_ind_1 = threadID + 1 + (s_rowwidth * 1);
    int s_ind_2 = threadID + 1 + (s_rowwidth * 2);
    int s_index_result = threadID + 1 + (s_rowwidth * 3);

    //Check the boundary. DO NOT copy data from out of bounds, but the thread MUST remain alive for syncthreads
    //Each thread should copy in 3 values
    if( i >= width - 1|| j >= width || i < 1 || j < 1 ) {
        // Do Nothing (Keep thread for __syncthreads)
    } else {
        s_data[s_ind_0-1] = g_dataA[g_ind_0-1];
        s_data[s_ind_1-1] = g_dataA[g_ind_1-1];
        s_data[s_ind_2-1] = g_dataA[g_ind_2-1];
        s_data[s_ind_0+1] = g_dataA[g_ind_0+1];
        s_data[s_ind_1+1] = g_dataA[g_ind_1+1];
        s_data[s_ind_2+1] = g_dataA[g_ind_2+1];
    }
    __syncthreads();

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) {
        // Do Nothing (Keep thread for __syncthreads)
    } else {
        const int s_i = 1;
        const int s_j = threadID + 1;
        const int pitch = s_rowwidth;

        //Calculate our cell's result using a LOCAL variable, then write that variable to the result
        s_data[s_index_result] = (
                                         0.2f * s_data[s_i * pitch + s_j] +               //itself
                                         0.1f * s_data[(s_i-1) * pitch +  s_j   ] +       //N
                                         0.1f * s_data[(s_i-1) * pitch + (s_j+1)] +       //NE
                                         0.1f * s_data[ s_i    * pitch + (s_j+1)] +       //E
                                         0.1f * s_data[(s_i+1) * pitch + (s_j+1)] +       //SE
                                         0.1f * s_data[(s_i+1) * pitch +  s_j   ] +       //S
                                         0.1f * s_data[(s_i+1) * pitch + (s_j-1)] +       //SW
                                         0.1f * s_data[ s_i    * pitch + (s_j-1)] +       //W
                                         0.1f * s_data[(s_i-1) * pitch + (s_j-1)]         //NW
                                 ) * 0.95f;
    }

    __syncthreads();

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) {
        return;
    }

    g_dataB[i * floatpitch + j] = s_data[s_index_result];
}
