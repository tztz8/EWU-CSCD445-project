#include "hip/hip_runtime.h"
// Logger Lib
#define SPDLOG_ACTIVE_LEVEL SPDLOG_LEVEL_TRACE
#include <spdlog/spdlog.h>
#include <GL/gl.h>

#include "cudaMain.cuh"

// Texture ID
GLuint cudaTexID;

double timeStart;

int device = 0;

float* h_dataA;
float* d_dataA;
float* d_dataB;
int width;
int height;
int threadsPerBlock;

GLuint genCudaTexImage() {
    GLuint tid;
    glGenTextures(1, &tid);
    glBindTexture(GL_TEXTURE_2D, tid);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0,  GL_RED,
                 GL_FLOAT, h_dataA);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);

    glBindTexture(GL_TEXTURE_2D, tid);
    return tid;
}

// Called when setting things up before graphs loop
__host__ void cudaMainInitialize(int size) {
    SPDLOG_INFO("Initialize Cuda");
    timeStart = 0;

    cudaTexID = genCudaTexImage();

}
__device__ int  get_x_lined(int * board, int col,int y,int x)
{
    return   board[(y * col) + x-1]+
             board[(y * col) + x]+
             board[(y * col) + x+1];
}

__device__ int  get_y_lined(int * board, int col,int y,int x)
{
    return   board[(y * col+1) + x]+
             board[(y * col) + x]+
             board[(y * col-1) + x];
}
__device__ int cudaAddUpLife(int * board,int col, int i, int j)
{
    return  get_x_lined(board, col, i+1,j)+
            get_x_lined(board, col, i,j)+
            get_x_lined(board, col, i-1,j);
}
__device__ int deadorAlive(int value, int currentLocation)
{
    if(currentLocation==1)
    {
        if(value-1==2||value-1==3)
        {
            return 1;
        }
        else
        {
            return 0;
        }
    }
    else
    {
        if(value==3)
        {
            return 1;
        }
        else
            return 0;
    }
}
//add WrapMain from CPU but should include to
__global__ void wrapMain( int *board, int *nextboard, int col, int row, int start, int wrapBoxColumn)
{
    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    int x;
    if(i < row - 1 && j < col - 1)
    {
        x = board[(j-1)*col+wrapBoxColumn]+
            board[(j-1)*col+start]+
            board[(j-1)*col+start+1]+
            board[(j)*col+wrapBoxColumn]+
            board[(j)*col+start]+
            board[(j)*col+start+1]+
            board[(j+1)*col+wrapBoxColumn]+
            board[(j+1)*col+start]+
            board[(j+1)*col+start+1];

        nextboard[j*col+start] = deadorAlive(x, board[j*col+start]);

        x = board[(j-1)*col+start] +
            board[(j-1)*col+wrapBoxColumn] +
            board[(j-1)*col+wrapBoxColumn - 1] +
            board[(j)*col+start] +
            board[(j)*col+wrapBoxColumn] +
            board[(j)*col+wrapBoxColumn - 1] +
            board[(j+1)*col+start] +
            board[(j+1)*col+wrapBoxColumn] +
            board[(j+1)*col+wrapBoxColumn - 1];
        nextboard[j*col+wrapBoxColumn] = deadorAlive(x, board[j*col+wrapBoxColumn]);
    }
    
}
__global__ void merge_top_front( int *board, int *nextboard, int col, int row, int start, int topstart)
{
    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    int x;
    x= get_x_lined(board,col,start,topstart+i)+
       get_x_lined(board,col,start,i)+
       get_x_lined(board,col,start+1,i);
    // y = start, x = i
    nextboard[start*col+i] = deadorAlive(x,board[start*col+i]);
    
}
__global__ void merge_top_back( int *board, int *nextboard, int col, int row, int start, int topstart, int wrapBoxRow, int backstart)
{
    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    int x;
    x= get_x_lined(board,col,wrapBoxRow,topstart+i)+
       get_x_lined(board,col,start,backstart+i)+
       get_x_lined(board,col,start+1,backstart+i);
    // y = start, x = backstart+i
    nextboard[start*col+(i+backstart)] = deadorAlive(x,board[start*col+(i+backstart)]);
}
//kept just in case it has usful info
__global__ void oldk1_from_hmwk( float* g_dataA, float* g_dataB, int floatpitch, int width)
{
    extern __shared__ float s_data[];
    //Write this kernel to achieve the same output as the provided k0, but you will have to use
    // shared memory.

    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    i = i + 1; //because the edge of the data is not processed

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    j = j + 1; //because the edge of the data is not processed

    int threadID = threadIdx.x;

    int s_rowwidth = blockDim.x + 2;

    // Index's

    // -- Global
    int g_ind_0 = (i-1) * floatpitch +  j;
    int g_ind_1 = i * floatpitch + j;
    int g_ind_2 = (i+1) * floatpitch +  j;

    // -- Shared
    int s_ind_0 = threadID + 1 + (s_rowwidth * 0);
    int s_ind_1 = threadID + 1 + (s_rowwidth * 1);
    int s_ind_2 = threadID + 1 + (s_rowwidth * 2);
    int s_index_result = threadID + 1 + (s_rowwidth * 3);

    //Check the boundary. DO NOT copy data from out of bounds, but the thread MUST remain alive for syncthreads
    //Each thread should copy in 3 values
    if( i >= width - 1|| j >= width || i < 1 || j < 1 ) {
        // Do Nothing (Keep thread for __syncthreads)
    } else {
        s_data[s_ind_0-1] = g_dataA[g_ind_0-1];
        s_data[s_ind_1-1] = g_dataA[g_ind_1-1];
        s_data[s_ind_2-1] = g_dataA[g_ind_2-1];
        s_data[s_ind_0+1] = g_dataA[g_ind_0+1];
        s_data[s_ind_1+1] = g_dataA[g_ind_1+1];
        s_data[s_ind_2+1] = g_dataA[g_ind_2+1];
    }
    __syncthreads();

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) {
        // Do Nothing (Keep thread for __syncthreads)
    } else {
        const int s_i = 1;
        const int s_j = threadID + 1;
        const int pitch = s_rowwidth;

        //Calculate our cell's result using a LOCAL variable, then write that variable to the result
        s_data[s_index_result] = (
                                         0.2f * s_data[s_i * pitch + s_j] +               //itself
                                         0.1f * s_data[(s_i-1) * pitch +  s_j   ] +       //N
                                         0.1f * s_data[(s_i-1) * pitch + (s_j+1)] +       //NE
                                         0.1f * s_data[ s_i    * pitch + (s_j+1)] +       //E
                                         0.1f * s_data[(s_i+1) * pitch + (s_j+1)] +       //SE
                                         0.1f * s_data[(s_i+1) * pitch +  s_j   ] +       //S
                                         0.1f * s_data[(s_i+1) * pitch + (s_j-1)] +       //SW
                                         0.1f * s_data[ s_i    * pitch + (s_j-1)] +       //W
                                         0.1f * s_data[(s_i-1) * pitch + (s_j-1)]         //NW
                                 ) * 0.95f;
    }

    __syncthreads();

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) {
        return;
    }

    g_dataB[i * floatpitch + j] = s_data[s_index_result];
}

// Called for every frame
__host__ void cudaMainUpdate() {
    hipError_t code = hipDeviceSynchronize();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Device Synchronize error -- {}", hipGetErrorString(code)));
    }
    // check if kernel execution generated an error
    code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }

    // copy result from device to host
    //hipMemcpy2D( h_dataA, width * sizeof(float), d_dataA, pitch, width * sizeof(float), height,hipMemcpyDeviceToHost );
    glDeleteTextures(1, &cudaTexID);
    cudaTexID = genCudaTexImage();

    //k1 <<< grid, threads, shared_mem_size >>>( d_dataA, d_dataB, pitch/sizeof(float), width);
}

// Called as the program close
__host__ void cudaMainCleanUp() {
    SPDLOG_INFO("Clean Up Cuda");

    // check if kernel execution generated an error
    hipError_t code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }
    // copy result from device to host
    //hipMemcpy2D( h_dataA, width * sizeof(float), d_dataA, pitch, width * sizeof(float), height,hipMemcpyDeviceToHost );

    free(h_dataA);
}
