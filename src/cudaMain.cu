#include "hip/hip_runtime.h"
// Logger Lib
#define SPDLOG_ACTIVE_LEVEL SPDLOG_LEVEL_TRACE
#include <spdlog/spdlog.h>
#include <GL/gl.h>

#include "cudaMain.cuh"
#include "string.h"
#include "stdlib.h"

int *h_board;
int *h_next_board;
int *d_board;
int *d_nextboard;
int rows;
int cols;
int size;

__device__ int get_x_lined(int * board, int col,int x,int y)
{
    return  (board[(y * col) + x-1])+
            (board[(y * col) + x])+
            (board[(y * col) + x+1]);
}

__device__ int cudaAddUpLife(int * board,int col, int x, int y)
{
    return  get_x_lined(board, col, x,y+1)+
            get_x_lined(board, col, x,y)+
            get_x_lined(board, col, x,y-1);
}
__device__ int deadorAlive(int value, int currentLocation)
{
    if (currentLocation == 1)
        if (value - 1 == 2 || value - 1 == 3)
            return 1;
        else
            return 0;
    else if (value == 3)
        return 1;
    else
        return 0;
}


__global__ void two_d_conway_block(int *board, int *next_board, int cols, int x_size, int y_size)
{
    // global thread(data) row index
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // global thread(data) column index
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    int index = y*cols+x;
    int total = 0;

    if((y < y_size) && (x < x_size)) {

        total = cudaAddUpLife(board,cols,x,y);
//        next_board[index] = deadorAlive(total,board[index]);

//        total += board[index-1-cols] + board[index-cols] + board[index + 1 - cols] +
//                 board[index - 1] + board[index] + board[index+1] +
//                 board[index + cols - 1] + board[index + cols] + board[index + cols +1];
        next_board[index] = (total == 3) || (total-(board[index]) == 3);
    }

}


void launch_two_d_conway_block(dim3 blocksize)
{
    const dim3 numBlocksMain((cols)/blocksize.x +1 ,rows/blocksize.y + 1,1);
    const dim3 numBlocks(size/blocksize.x + 1,size/blocksize.y + 1,1);
    two_d_conway_block<<<numBlocksMain, blocksize>>>(d_board + 1 + cols, d_nextboard + 1 + cols, cols, (size*4)-2, size-2);
    two_d_conway_block<<<numBlocks, blocksize>>>(d_board + 1 + size*4 + cols, d_nextboard + 1 + size*4 + cols, cols, size-2, size-2);
    two_d_conway_block<<<numBlocks, blocksize>>>(d_board + 1 + size*5 + cols, d_nextboard + 1 + size*5 + cols, cols, size-2, size-2);
}


__global__ void conway_edges(int *self, int *self_next, int *other, int self_idx_step, int other_idx_step, int adjacent_offset, int len) {
    //Determine the index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx>=len) return;

    //Rebase the indexes for this thread
    self = self + self_idx_step*idx;
    other = other + other_idx_step*idx;
    self_next = self_next + self_idx_step*idx;

    //Get the complete sum
    int sum =   *(other - other_idx_step) + (*other) + *(other + other_idx_step) +
                *(self  - self_idx_step ) + (*self ) + *(self  + self_idx_step ) +
                *(self + adjacent_offset - self_idx_step) + *(self + adjacent_offset) + *(self + adjacent_offset + self_idx_step);

    //Write the result
    *self_next = (sum == 3) || (sum-(*self) == 3);

}

void cuda_launch_conway_edges(int blocksize) {
    const int edge_length = size - 2;
    const dim3 numBlocks(edge_length/blocksize + (1&&(edge_length%blocksize)), 1, 1);

    //wrap
    conway_edges<<<numBlocks, blocksize>>>(d_board + cols, d_nextboard + cols, d_board + size*4 - 1 + cols, cols, cols, 1, edge_length); // front to left
    conway_edges<<<numBlocks, blocksize>>>(d_board + size*4 - 1 + cols, d_nextboard + size*4 - 1 + cols, d_board + cols, cols, cols, -1, edge_length); //left to front

    //start of top
    conway_edges <<<numBlocks, blocksize>>>(d_board + 1, d_nextboard + 1, d_board + (4 * size)+1, 1, 1, cols, edge_length); //top to front
    conway_edges <<<numBlocks, blocksize>>>(d_board + (4 * size)+1, d_nextboard + (4 * size)+1, d_board + 1, 1, 1, cols, edge_length); //front to top

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size*2, d_nextboard + 1 + size*2, d_board + 5*size - 2 + cols*(rows-1), 1, -1, cols, edge_length); //top to back
    conway_edges <<<numBlocks, blocksize>>>(d_board + 5*size - 2 + cols*(rows-1), d_nextboard + 5*size - 2 + cols*(rows-1), d_board + 1 + size*2, -1, 1, -cols, edge_length); //back to top

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size, d_nextboard + 1 + size, d_board + 5*size - 1 + cols, 1, cols, cols, edge_length); //top to right
    conway_edges <<<numBlocks, blocksize>>>(d_board + 5*size - 1 + cols, d_nextboard + 5*size - 1 + cols, d_board + 1 + size, cols, 1, -1, edge_length); //right to top

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size*3, d_nextboard + 1 + size*3, d_board + 4*size + cols*(rows-2), 1, (-cols), cols, edge_length); //top to left
    conway_edges <<<numBlocks, blocksize>>>(d_board + 4*size + cols*(rows-2), d_nextboard + 4*size + cols*(rows-2), d_board + 1 + size*3, (-cols), 1, 1, edge_length); //left to top

    //start of bottom
    conway_edges <<<numBlocks, blocksize>>>(d_board+cols*(rows-1)+1, d_nextboard+cols*(rows-1)+1, d_board + size*5 + 1, 1, 1, (-cols), edge_length); // bottom to front
    conway_edges <<<numBlocks, blocksize>>>(d_board + size*5 + 1, d_nextboard + size*5 + 1, d_board+cols*(rows-1)+1, 1, 1, (-cols), edge_length); // front to bottom

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size*2 + cols*(rows-1), d_nextboard + 1 + size*2 + cols*(rows-1), d_board + size*6 - 2 + cols*(rows-1), 1, -1, (-cols), edge_length); // bottom to back
    conway_edges <<<numBlocks, blocksize>>>(d_board + size*6 - 2 + cols*(rows-1), d_nextboard + size*6 - 2 + cols*(rows-1), d_board + 1 + size*2 + cols*(rows-1), -1, 1, (-cols), edge_length); // back to bottom

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size + cols*(rows-1), d_nextboard + 1 + size + cols*(rows-1), d_board + size*6 - 1 + cols, 1, cols, (-cols), edge_length); //bottom to right
    conway_edges <<<numBlocks, blocksize>>>(d_board + size*6 - 1 + cols, d_nextboard + size*6 - 1 + cols, d_board + 1 + size + cols*(rows-1), cols, 1, -1, edge_length); //right to bottom

    conway_edges <<<numBlocks, blocksize>>>(d_board + size*3 + 1 + cols*(rows-1), d_nextboard + size*3 + 1 + cols*(rows-1), d_board + size*5 + cols*(rows-2), 1, (-cols), (-cols), edge_length); // bottom to left
    conway_edges <<<numBlocks, blocksize>>>(d_board + size*5 + cols*(rows-2), d_nextboard + size*5 + cols*(rows-2), d_board + size*3 + 1 + cols*(rows-1), (-cols), 1, 1, edge_length); // bottom to left
}

// Called when setting things up before graphs loop
__host__ void cudaMainInitialize(int size_set) {
    SPDLOG_INFO("Initialize Cuda");
    float timeStart = 0;
    size = size_set;
    rows = size;
    cols = size*6;

    h_board = (int *) calloc(rows*cols, sizeof(int));
    h_next_board = (int *) calloc(rows*cols, sizeof(int));

    //Initialize a pattern in the conway grid
    for (int i = 0; i < cols; ++i) {
        h_board[(3 * cols) + i] = 1;
    }
// Glider
//    h_board[3*cols + 110] = 1;
//    h_board[3*cols + 111] = 1;
//    h_board[3*cols + 112] = 1;
//    h_board[4*cols + 110] = 1;
//    h_board[5*cols + 111] = 1;
//
//    h_board[3*cols + 110 + size] = 1;
//    h_board[3*cols + 111 + size] = 1;
//    h_board[3*cols + 112 + size] = 1;
//    h_board[4*cols + 110 + size] = 1;
//    h_board[5*cols + 111 + size] = 1;

    //cuda memory allocation
    hipMalloc(&d_board, sizeof(int)*rows*cols);
    hipMalloc(&d_nextboard, sizeof(int)*rows*cols);

    hipMemcpy(d_board, h_board, sizeof(int)*rows*cols, hipMemcpyHostToDevice);
}

// Called for every frame
int *cudaMainUpdate() {
    hipError_t code = hipDeviceSynchronize();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Device Synchronize error -- {}", hipGetErrorString(code)));
    }
    // check if kernel execution generated an error
    code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }
    const int blocksize = 32;
    const dim3 blockdim(blocksize, blocksize, 1);

    memcpy(h_next_board, h_board, rows*cols*sizeof(*h_board));

    hipMemcpy(d_nextboard, h_next_board, rows*cols*sizeof(*d_nextboard), hipMemcpyHostToDevice);
    hipMemcpy(d_board, h_board, rows*cols*sizeof(*d_nextboard), hipMemcpyHostToDevice);

    // #### Field ####
    launch_two_d_conway_block(blockdim);

    // #### Edges ####
    const int edge_length = size - 2;
    const dim3 numBlocks(edge_length/blocksize + (1&&(edge_length%blocksize)), 1, 1);

    cuda_launch_conway_edges(blocksize);

    hipMemcpy(h_board, d_nextboard, rows*cols*sizeof(*d_nextboard), hipMemcpyDeviceToHost);
    {
        int * temp = d_board;
        d_board = d_nextboard;
        d_nextboard = temp;
    }
    return h_board;
}

// Called as the program close
__host__ void cudaMainCleanUp() {
    SPDLOG_INFO("Clean Up Cuda");

    // check if kernel execution generated an error
    hipError_t code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }

    hipFree(d_board);
    hipFree(d_nextboard);
    free(h_board);
    free(h_next_board);
}