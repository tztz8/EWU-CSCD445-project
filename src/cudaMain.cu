#include "hip/hip_runtime.h"
// Logger Lib
#define SPDLOG_ACTIVE_LEVEL SPDLOG_LEVEL_TRACE
#include <spdlog/spdlog.h>

#include "cudaMain.cuh"

double timeStart;

int device = 0;

float* h_dataA;
float* d_dataA;
float* d_dataB;
int width;
int height;
int threadsPerBlock;

size_t pitch;

unsigned int shared_mem_size;
dim3  grid;
dim3  threads;

void initializeArrays(float *a1, float *a2, int width, int height){
    int i, j;

    for(i=0; i<height; i++){
        for(j=0; j<width; j++){
            if(i==0 || j ==0 || i==height-1 || j==width-1){
                a1[i*width + j] = 5.0;
                a2[i*width + j] = 5.0;
            } else {
                a1[i*width + j] = 1.0;
            }
        }
    }
}

// Called when setting things up before graphs loop
__host__ void cudaMainInitialize() {
    SPDLOG_INFO("Initialize Cuda");
    timeStart = 0;

    threadsPerBlock = 32;
    width = 3200;
    height = 3200;

    h_dataA= (float *)malloc(width * height * sizeof(float));
    float * h_dataB= (float *)malloc(width * height * sizeof(float));

    initializeArrays(h_dataA, h_dataB, width, height);

    // Use card 0  (See top of file to make sure you are using your assigned device.)
    hipSetDevice(device);

    // To ensure alignment, we'll use the code below to pad rows of the arrays when they are
    // allocated on the device.
    // allocate device memory for data A
    hipMallocPitch( (void**) &d_dataA, &pitch, width * sizeof(float), height);

    // copy host memory to device memory for image A
    hipMemcpy2D( d_dataA, pitch, h_dataA, width * sizeof(float), width * sizeof(float), height,
                  hipMemcpyHostToDevice);

    // repeat for second device array
    hipMallocPitch( (void**) &d_dataB, &pitch, width * sizeof(float), height);

    // copy host memory to device memory for image B
    hipMemcpy2D( d_dataB, pitch, h_dataB, width * sizeof(float), width * sizeof(float), height,
                  hipMemcpyHostToDevice) ;

    //***************************
    // setup CUDA execution parameters

    int blockHeight;
    int blockWidth;

    // When testing with small arrays, this code might be useful. Feel free to change it.
    if (threadsPerBlock > width - 2 ){
        blockWidth = 16 * (int) ceil((width - 2) / 16.0);
        blockHeight = 1;
    } else {

        blockWidth = threadsPerBlock;
        blockHeight = 1;
    }

    int gridWidth = (int) ceil( (width - 2) / (float) blockWidth);
    int gridHeight = (int) ceil( (height - 2) / (float) blockHeight);

    // number of blocks required to process all the data.
    int numBlocks =   gridWidth * gridHeight;

    // Each block gets a shared memory region of this size.
    shared_mem_size = ((blockWidth + 2) * 4) * sizeof(float);

    SPDLOG_INFO(spdlog::fmt_lib::format("blockDim.x={} blockDim.y={}    grid = {} x {}", blockWidth, blockHeight, gridWidth, gridHeight));
    SPDLOG_INFO(spdlog::fmt_lib::format("numBlocks = {},  threadsPerBlock = {}   shared_mem_size = {}", numBlocks, threadsPerBlock,  shared_mem_size));

    if(gridWidth > 65536 || gridHeight > 65536) {
        SPDLOG_ERROR("a block dimension is too large.");
    }

    if(threadsPerBlock > 1024) {
        SPDLOG_ERROR("number of threads per block is too large.");
    }

    if(shared_mem_size > 49152) {
        SPDLOG_ERROR("shared memory per block is too large.");
    }

    // Format the grid, which is a collection of blocks.
    grid = dim3( gridWidth, gridHeight, 1);

    // Format the blocks.
    threads = dim3( blockWidth, blockHeight, 1);


    // time the kernel launches using CUDA events
    hipEvent_t launch_begin, launch_end;
    hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);

    hipEventRecord(launch_begin,0);
}

__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width)
{
    extern __shared__ float s_data[];
    //Write this kernel to achieve the same output as the provided k0, but you will have to use
    // shared memory.

    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    i = i + 1; //because the edge of the data is not processed

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    j = j + 1; //because the edge of the data is not processed

    int threadID = threadIdx.x;

    int s_rowwidth = blockDim.x + 2;

    // Index's

    // -- Global
    int g_ind_0 = (i-1) * floatpitch +  j;
    int g_ind_1 = i * floatpitch + j;
    int g_ind_2 = (i+1) * floatpitch +  j;

    // -- Shared
    int s_ind_0 = threadID + 1 + (s_rowwidth * 0);
    int s_ind_1 = threadID + 1 + (s_rowwidth * 1);
    int s_ind_2 = threadID + 1 + (s_rowwidth * 2);
    int s_index_result = threadID + 1 + (s_rowwidth * 3);

    //Check the boundary. DO NOT copy data from out of bounds, but the thread MUST remain alive for syncthreads
    //Each thread should copy in 3 values
    if( i >= width - 1|| j >= width || i < 1 || j < 1 ) {
        // Do Nothing (Keep thread for __syncthreads)
    } else {
        s_data[s_ind_0-1] = g_dataA[g_ind_0-1];
        s_data[s_ind_1-1] = g_dataA[g_ind_1-1];
        s_data[s_ind_2-1] = g_dataA[g_ind_2-1];
        s_data[s_ind_0+1] = g_dataA[g_ind_0+1];
        s_data[s_ind_1+1] = g_dataA[g_ind_1+1];
        s_data[s_ind_2+1] = g_dataA[g_ind_2+1];
    }
    __syncthreads();

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) {
        // Do Nothing (Keep thread for __syncthreads)
    } else {
        const int s_i = 1;
        const int s_j = threadID + 1;
        const int pitch = s_rowwidth;

        //Calculate our cell's result using a LOCAL variable, then write that variable to the result
        s_data[s_index_result] = (
                                         0.2f * s_data[s_i * pitch + s_j] +               //itself
                                         0.1f * s_data[(s_i-1) * pitch +  s_j   ] +       //N
                                         0.1f * s_data[(s_i-1) * pitch + (s_j+1)] +       //NE
                                         0.1f * s_data[ s_i    * pitch + (s_j+1)] +       //E
                                         0.1f * s_data[(s_i+1) * pitch + (s_j+1)] +       //SE
                                         0.1f * s_data[(s_i+1) * pitch +  s_j   ] +       //S
                                         0.1f * s_data[(s_i+1) * pitch + (s_j-1)] +       //SW
                                         0.1f * s_data[ s_i    * pitch + (s_j-1)] +       //W
                                         0.1f * s_data[(s_i-1) * pitch + (s_j-1)]         //NW
                                 ) * 0.95f;
    }

    __syncthreads();

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) {
        return;
    }

    g_dataB[i * floatpitch + j] = s_data[s_index_result];
}

// Called for every frame
__host__ void cudaMainUpdate(double time) {
    bool error = false;
    if (error) {
        // Avoid logging messages
        SPDLOG_ERROR("Problem Happen");
    }
//    if (time-timeStart > 0.5F) {
//        SPDLOG_INFO("Start New CUDA Kernel");
//        timeStart = time;
//        hipError_t code = hipDeviceSynchronize();
//        if (code != hipSuccess){
//            SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Device Synchronize error -- {}", hipGetErrorString(code)));
//        }
//        // check if kernel execution generated an error
//        code = hipGetLastError();
//        if (code != hipSuccess){
//            SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
//        }
//        k1 <<< grid, threads, shared_mem_size >>>( d_dataA, d_dataB, pitch/sizeof(float), width);
//    }
    hipError_t code = hipDeviceSynchronize();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Device Synchronize error -- {}", hipGetErrorString(code)));
    }
    // check if kernel execution generated an error
    code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }
    k1 <<< grid, threads, shared_mem_size >>>( d_dataA, d_dataB, pitch/sizeof(float), width);
}

// Called as the program close
__host__ void cudaMainCleanUp() {
    SPDLOG_INFO("Clean Up Cuda");

    // check if kernel execution generated an error
    hipError_t code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }
    // copy result from device to host
    hipMemcpy2D( h_dataA, width * sizeof(float), d_dataA, pitch, width * sizeof(float), height,hipMemcpyDeviceToHost );
}
