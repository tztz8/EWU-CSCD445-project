#include "hip/hip_runtime.h"
// Logger Lib
#define SPDLOG_ACTIVE_LEVEL SPDLOG_LEVEL_TRACE
#include <spdlog/spdlog.h>
#include <GL/gl.h>

#include "cudaMain.cuh"

// Texture ID
GLuint cudaTexID;

double timeStart;

int device = 0;

float* h_dataA;
float* d_dataA;
float* d_dataB;
int width;
int height;
int threadsPerBlock;

GLuint genCudaTexImage() {
    GLuint tid;
    glGenTextures(1, &tid);
    glBindTexture(GL_TEXTURE_2D, tid);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0,  GL_RED,
                 GL_FLOAT, h_dataA);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);

    glBindTexture(GL_TEXTURE_2D, tid);
    return tid;
}

// Called when setting things up before graphs loop
__host__ void cudaMainInitialize(int size) {
    SPDLOG_INFO("Initialize Cuda");
    timeStart = 0;

    threadsPerBlock = 32;
    width = size * 6;
    height = size;

    h_dataA= (float *)malloc(width * height * sizeof(float));
    float * h_dataB= (float *)malloc(width * height * sizeof(float));

    initializeArrays(h_dataA, h_dataB, width, height);

    cudaTexID = genCudaTexImage();

    // Use card 0  (See top of file to make sure you are using your assigned device.)
    hipSetDevice(device);

    // To ensure alignment, we'll use the code below to pad rows of the arrays when they are
    // allocated on the device.
    // allocate device memory for data A
    hipMallocPitch( (void**) &d_dataA, &pitch, width * sizeof(float), height);

    // copy host memory to device memory for image A
    hipMemcpy2D( d_dataA, pitch, h_dataA, width * sizeof(float), width * sizeof(float), height,
                  hipMemcpyHostToDevice);

    // repeat for second device array
    hipMallocPitch( (void**) &d_dataB, &pitch, width * sizeof(float), height);

    // copy host memory to device memory for image B
    hipMemcpy2D( d_dataB, pitch, h_dataB, width * sizeof(float), width * sizeof(float), height,
                  hipMemcpyHostToDevice) ;

    free(h_dataB);

    //***************************
    // setup CUDA execution parameters

    int blockHeight;
    int blockWidth;

    // When testing with small arrays, this code might be useful. Feel free to change it.
    if (threadsPerBlock > width - 2 ){
        blockWidth = 16 * (int) ceil((width - 2) / 16.0);
        blockHeight = 1;
    } else {

        blockWidth = threadsPerBlock;
        blockHeight = 1;
    }

    int gridWidth = (int) ceil( (width - 2) / (float) blockWidth);
    int gridHeight = (int) ceil( (height - 2) / (float) blockHeight);

    // number of blocks required to process all the data.
    int numBlocks =   gridWidth * gridHeight;

    // Each block gets a shared memory region of this size.
    shared_mem_size = ((blockWidth + 2) * 4) * sizeof(float);

    SPDLOG_INFO(spdlog::fmt_lib::format("blockDim.x={} blockDim.y={}    grid = {} x {}", blockWidth, blockHeight, gridWidth, gridHeight));
    SPDLOG_INFO(spdlog::fmt_lib::format("numBlocks = {},  threadsPerBlock = {}   shared_mem_size = {}", numBlocks, threadsPerBlock,  shared_mem_size));

    if(gridWidth > 65536 || gridHeight > 65536) {
        SPDLOG_ERROR("a block dimension is too large.");
    }

    if(threadsPerBlock > 1024) {
        SPDLOG_ERROR("number of threads per block is too large.");
    }

    if(shared_mem_size > 49152) {
        SPDLOG_ERROR("shared memory per block is too large.");
    }

    // Format the grid, which is a collection of blocks.
    grid = dim3( gridWidth, gridHeight, 1);

    // Format the blocks.
    threads = dim3( blockWidth, blockHeight, 1);


    // time the kernel launches using CUDA events
    hipEvent_t launch_begin, launch_end;
    hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);

    hipEventRecord(launch_begin,0);
}

__device__ int deadorAlive(int value, int current)
{
    if(current==1)
    {
        if(value-1==2||value-1==3)
        {
            return 1;
        }
        else
        {
            return 0;
        }
    }
    else
    {
        if(value==3)
        {
            return 1;
        }
        else
            return 0;
    }
}
//add WrapMain from CPU
__global__ void k1( int *board, int *nextboard, int col, int row, int start, int wrapBoxColumn)
{
    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    int x;
    if(i < row - 1 && j < col - 1)
    {
        x = board[(j-1)*col+wrapBoxColumn]+
            board[(j-1)*col+start]+
            board[(j-1)*col+start+1]+
            board[(j)*col+wrapBoxColumn]+
            board[(j)*col+start]+
            board[(j)*col+start+1]+
            board[(j+1)*col+wrapBoxColumn]+
            board[(j+1)*col+start]+
            board[(j+1)*col+start+1];

        nextboard[i*col+x] = deadorAlive(x, board[i*col+x]);

        x = board[(j-1)*col+start] +
            board[(j-1)*col+wrapBoxColumn] +
            board[(j-1)*col+wrapBoxColumn - 1] +
            board[(j)*col+start] +
            board[(j)*col+wrapBoxColumn] +
            board[(j)*col+wrapBoxColumn - 1] +
            board[(j+1)*col+start] +
            board[(j+1)*col+wrapBoxColumn] +
            board[(j+1)*col+wrapBoxColumn - 1];
        nextboard[i*col+x] = deadorAlive(x, board[i*col+x]);
    }
    
}
//kept just in case it has usful info
__global__ void oldk1_from_hmwk( float* g_dataA, float* g_dataB, int floatpitch, int width)
{
    extern __shared__ float s_data[];
    //Write this kernel to achieve the same output as the provided k0, but you will have to use
    // shared memory.

    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    i = i + 1; //because the edge of the data is not processed

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    j = j + 1; //because the edge of the data is not processed

    int threadID = threadIdx.x;

    int s_rowwidth = blockDim.x + 2;

    // Index's

    // -- Global
    int g_ind_0 = (i-1) * floatpitch +  j;
    int g_ind_1 = i * floatpitch + j;
    int g_ind_2 = (i+1) * floatpitch +  j;

    // -- Shared
    int s_ind_0 = threadID + 1 + (s_rowwidth * 0);
    int s_ind_1 = threadID + 1 + (s_rowwidth * 1);
    int s_ind_2 = threadID + 1 + (s_rowwidth * 2);
    int s_index_result = threadID + 1 + (s_rowwidth * 3);

    //Check the boundary. DO NOT copy data from out of bounds, but the thread MUST remain alive for syncthreads
    //Each thread should copy in 3 values
    if( i >= width - 1|| j >= width || i < 1 || j < 1 ) {
        // Do Nothing (Keep thread for __syncthreads)
    } else {
        s_data[s_ind_0-1] = g_dataA[g_ind_0-1];
        s_data[s_ind_1-1] = g_dataA[g_ind_1-1];
        s_data[s_ind_2-1] = g_dataA[g_ind_2-1];
        s_data[s_ind_0+1] = g_dataA[g_ind_0+1];
        s_data[s_ind_1+1] = g_dataA[g_ind_1+1];
        s_data[s_ind_2+1] = g_dataA[g_ind_2+1];
    }
    __syncthreads();

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) {
        // Do Nothing (Keep thread for __syncthreads)
    } else {
        const int s_i = 1;
        const int s_j = threadID + 1;
        const int pitch = s_rowwidth;

        //Calculate our cell's result using a LOCAL variable, then write that variable to the result
        s_data[s_index_result] = (
                                         0.2f * s_data[s_i * pitch + s_j] +               //itself
                                         0.1f * s_data[(s_i-1) * pitch +  s_j   ] +       //N
                                         0.1f * s_data[(s_i-1) * pitch + (s_j+1)] +       //NE
                                         0.1f * s_data[ s_i    * pitch + (s_j+1)] +       //E
                                         0.1f * s_data[(s_i+1) * pitch + (s_j+1)] +       //SE
                                         0.1f * s_data[(s_i+1) * pitch +  s_j   ] +       //S
                                         0.1f * s_data[(s_i+1) * pitch + (s_j-1)] +       //SW
                                         0.1f * s_data[ s_i    * pitch + (s_j-1)] +       //W
                                         0.1f * s_data[(s_i-1) * pitch + (s_j-1)]         //NW
                                 ) * 0.95f;
    }

    __syncthreads();

    if( i >= width - 1|| j >= width - 1 || i < 1 || j < 1 ) {
        return;
    }

    g_dataB[i * floatpitch + j] = s_data[s_index_result];
}

// Called for every frame
__host__ void cudaMainUpdate() {
    hipError_t code = hipDeviceSynchronize();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Device Synchronize error -- {}", hipGetErrorString(code)));
    }
    // check if kernel execution generated an error
    code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }

    // copy result from device to host
    hipMemcpy2D( h_dataA, width * sizeof(float), d_dataA, pitch, width * sizeof(float), height,hipMemcpyDeviceToHost );
    glDeleteTextures(1, &cudaTexID);
    cudaTexID = genCudaTexImage();

    //k1 <<< grid, threads, shared_mem_size >>>( d_dataA, d_dataB, pitch/sizeof(float), width);
}

// Called as the program close
__host__ void cudaMainCleanUp() {
    SPDLOG_INFO("Clean Up Cuda");

    // check if kernel execution generated an error
    hipError_t code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }
    // copy result from device to host
    hipMemcpy2D( h_dataA, width * sizeof(float), d_dataA, pitch, width * sizeof(float), height,hipMemcpyDeviceToHost );

    free(h_dataA);
}
