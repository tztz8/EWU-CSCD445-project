#include "hip/hip_runtime.h"
// Logger Lib
#define SPDLOG_ACTIVE_LEVEL SPDLOG_LEVEL_TRACE
#include <spdlog/spdlog.h>
#include <GL/gl.h>

#include "cudaMain.cuh"
#include "string.h"
#include "stdlib.h"

int *h_board;
int *h_next_board;
int *d_board;
int *d_nextboard;
int rows;
int cols;
int size;

__host__ int  cpu_get_x_lined(int * board, int col,int y,int x)
{
    return   board[(y * col) + x-1]+
             board[(y * col) + x]+
             board[(y * col) + x+1];
}

__host__ int cpuDeadorAlive(int value, int currentLocation){
    if (currentLocation == 1)
        if (value - 1 == 2 || value - 1 == 3)
            return 1;
        else
            return 0;
    else if (value == 3)
        return 1;
    else
        return 0;
}

// the y and x values should be the max-1 and are static for this
// function

//TODO need to check if the y needs to be "-1"
__host__ void cornerHost(int* h_board, int * h_next_board,int y, int x)
{
    int *board =h_board;
    int *nextboard = h_next_board;
    int value=0;
    y--;
    // the first four or the corners of the main box (warps)
    //left corner of front to top and left (warps)
    value = board[((x/3)*2)]+
            board[((x/3)*2)+1]+
            board[4*(x/6)-1]+
            board[0]+
            board[1]+
            board[(1)*x+(4*(x/6)-1)]+
            board[(1)*x+0]+
            board[(1)*x+0+1];

    nextboard[0]= cpuDeadorAlive(value,board[0]);

    // Right top corner of Left to top and front (warps)
    value = board[x*(y-1)+((x/3)*2)]+
            board[x*(y)+((x/3)*2)]+

            board[0*x+(4*(x/6)-1)-1]+
            board[0*x+(4*(x/6)-1)]+
            board[0*x+0]+

            board[(0+1)*x+(4*(x/6)-1)-1]+
            board[(0+1)*x+(4*(x/6)-1)]+
            board[(0+1)*x+0+1];

    nextboard[0*x+(4*(x/6)-1)]= cpuDeadorAlive(value,board[0*x+(4*(x/6)-1)]);

    // left bot corner of front with left and bot (warps)

    value = board[(y-1)*x+(4*(x/6)-1)]+
            board[(y-1)*x+((x/3)*2)]+
            board[(y-1)*x+((x/3)*2)+1]+

            board[y*x+(4*(x/6)-1)]+
            board[y*x+0]+
            board[y*x+0+1]+

            board[0*x+((x/6)*5)]+
            board[0*x+((x/6)*5)+1];

            nextboard[y*x+0]= cpuDeadorAlive(value,
                                             board[y*x+0]);
    // right bot corner of left with top and bot (warps)
    value = board[(y-1)*x+(4*(x/6)-1)-1]+
            board[(y-1)*x+(4*(x/6)-1)]+
            board[(y-1)*x+0]+

            board[(y)*x+(4*(x/6)-1)-1]+
            board[(y)*x+(4*(x/6)-1)]+
            board[(y)*x+0]+

            board[0*x+((x/6)*5)]+
            board[(0+1)*x+((x/6)*5)];

    nextboard[(y)*x+(4*(x/6)-1)]= cpuDeadorAlive(value,
                                     board[(y)*x+(4*(x/6)-1)]);

    // left top corners with no wraps
    // so the next set are all left top corners
    // right with front and top
    value =     board[0*x+((x/6)*5-1)]+
            board[(0+1)*x+((x/6)*5-1)]+
            cpu_get_x_lined(board,x,0,(x/6))+
            cpu_get_x_lined(board,x,1,(x/6));

    nextboard[0*x+(x/6)]= cpuDeadorAlive(value,
                                         board[0*x+(x/6)]);

    // back with right and top
    value =     board[y*x+((x/3)*2)]+
                board[y*x+((x/3)*2)+1]+

    cpu_get_x_lined(board,x,0,(2*(x/6)))+
    cpu_get_x_lined(board,x,1,(2*(x/6)));

    nextboard[0*x+(2*(x/6))]= cpuDeadorAlive(value,
                                         board[0*x+(2*(x/6))]);

    // left with back and top
    value = board[0*x+((x/3)*2)]+
            board[(0+1)*x+((x/3)*2)]+

    cpu_get_x_lined(board,x,0,(3*(x/6)))+
    cpu_get_x_lined(board,x,1,(3*(x/6)));

    nextboard[0*x+(3*(x/6))]= cpuDeadorAlive(value,
                                             board[0*x+(3*(x/6))]);

    // the next set is the right top corners
    // front with right and top
    value =     board[0*x+((x/6)*5-1)]+
            board[0*x+((x/6)*5-1)-1]+

    cpu_get_x_lined(board,x,0,(x/6-1))+
    cpu_get_x_lined(board,x,1,(x/6-1));

    nextboard[0*x+(x/6-1)]= cpuDeadorAlive(value,
                                             board[0*x+(x/6-1)]);

    // right with top and back
    value =     board[y*x+((x/6)*5-1)]+
            board[(y-1)*x+((x/6)*5-1)]+

    cpu_get_x_lined(board,x,0,(2*(x/6)-1))+
    cpu_get_x_lined(board, x,0+1,(2*(x/6)-1));

    nextboard[0*x+(2*(x/6)-1)]= cpuDeadorAlive(value,
                                           board[0*x+(2*(x/6)-1)]);


    // right corner back
    value =     board[y*x+((x/6)*5-1)]+
            board[y*x+((x/6)*5-1)-1]+

    cpu_get_x_lined(board,x,0,(3*(x/6)-1))+
    cpu_get_x_lined(board,x,0+1,(3*(x/6)-1));

    nextboard[0*x+(2*(x/6)-1)]= cpuDeadorAlive(value,
                                               board[0*x+(3*(x/6)-1)]);

    // the next set handles the left bots corners
    // right with front and bot
    value = board[0*x+(x - 1)]+
            board[0*x+(x - 1)-1]+

    cpu_get_x_lined(board,x,y,(x/6))+
    cpu_get_x_lined(board,x,y-1,(x/6));

    nextboard[y*x+(x/6-1)]= cpuDeadorAlive(value,
                                               board[y*x+(x/6-1)]);

    // back with right and bot
    value = board[(y*x) + (x - 1)]+
            board[((y-1)*x) + (x - 1)]+

    cpu_get_x_lined(board,x,y,(2*(x/6)))+
    cpu_get_x_lined(board,x,y-1,(2*(x/6)));

    nextboard[y*x+(2*(x/6)-1)]= cpuDeadorAlive(value,
                                           board[y*x+(2*(x/6)-1)]);
    // left with back and bot
    value =     board[y*x+((x/6)*5)]+
            board[(y-1)*x+((x/6)*5)+1]+

    cpu_get_x_lined(board,x,y,(3*(x/6)))+
    cpu_get_x_lined(board,x,y-1,(3*(x/6)));

    nextboard[y*x+(3*(x/6))]= cpuDeadorAlive(value,
                                               board[y*x+(3*(x/6))]);

    // bot right corners of main
    // front with right and bot

    value = board[0*x+(5*(x/6))]+
    board[(0+1)*x+(5*(x/6))]+

    cpu_get_x_lined(board,x,y,(x/6))+
    cpu_get_x_lined(board,x,y-1,(x/6));

    nextboard[y*x+(x/6)]= cpuDeadorAlive(value,
                                             board[y*x+(x/6)]);

    // right with back and bot
    value =     board[y*x+(x - 1)]+
            board[(y)*x+(x - 1)-1]+

    cpu_get_x_lined(board,x,y,2*(x/6)-1)+
    cpu_get_x_lined(board,x,y-1,2*(x/6)-1);

    nextboard[y*x+2*(x/6)-1]= cpuDeadorAlive(value,
                                         board[y*x+2*(x/6)-1]);

    // back with left and bot
    value =     board[y*x+5*(x/6)]+
            board[y*x+5*(x/6)+1]+

    cpu_get_x_lined(board,x,y,(3*(x/6)-1))+
    cpu_get_x_lined(board,x,y-1,(3*(x/6)-1));

    nextboard[y*x+(3*(x/6)-1)]= cpuDeadorAlive(value,
                                             board[y*x+(3*(x/6)-1)]);

// For the top
// top with left and front

    value   =   board[0*x+0]+
                board[0*x+0+1]+

                board[0*x+(3*(x/6))]+
                board[0*x+((x/3)*2)]+
                board[0*x+((x/3)*2)+1]+

                board[(1)*x+(3*(x/6))]+
                board[(1)*x+((x/3)*2)]+
                board[(1)*x+((x/3)*2)+1];

    nextboard[0*x+((x/3)*2)]= cpuDeadorAlive(value,
                                               board[0*x+((x/3)*2)]);

    //top with right and front
    value =     board[0*x+(x/6-1)-1]+
            board[0*x+(x/6-1)]+

            board[0*x+((x/6)*5-1)-1]+
            board[0*x+((x/6)*5-1)]+
            board[0*x+(x/6)]+

            board[(1)*x+((x/6)*5-1)-1]+
            board[(1)*x+((x/6)*5-1)]+
            board[(0)*x+(x/6)+1];

    nextboard[0*x+((x/6)*5-1)]= cpuDeadorAlive(value,
                                             board[0*x+((x/6)*5-1)]);

    //Top with back and right
    value = board[0*x+(2*(x/6))]+
            board[0*x+(2*(x/6))+1];

            board[0*x+(2*(x/6)-1)]+
            board[y*x+((x/6)*5-1)]+
            board[y*x+((x/6)*5-1)+1]+

            board[1*x+(2*(x/6)-1)]+
            board[(y-1)*x+((x/6)*5-1)]+
            board[(y-1)*x+((x/6)*5-1)-1];

            nextboard[y*x+((x/6)*5-1)] = cpuDeadorAlive(value,
                                                       board[y*x+((x/6)*5-1)]);


    // Top with back left
    value =  board[0*x+(3*(x/6)-1)]+
             board[0*x+(3*(x/6)-1)-1]+

            board[0*x+(3*(x/6))]+
            board[(y)*x+((x/3)*2)]+
            board[(y)*x+((x/3)*2)+1]+

            board[0*x+(3*(x/6))+1]+
            board[(y-1)*x+((x/3)*2)]+
            board[(y-1)*x+((x/3)*2)+1];

    nextboard[(y)*x+((x/3)*2)] = cpuDeadorAlive(value,
                                                board[(y)*x+((x/3)*2)]);


    //botmerge corner
    // bot with front and left
        value = board[y*x]+
                board[y*x+1]+

                board[y*x+(4*(x/6)-1)]+
                board[0*x+((x/6)*5)]+
                board[0*x+((x/6)*5)+1]+

                //was wrapBoxRow+1
                board[(y) * x + (4*(x/6))]+
                board[(1)*x+((x/6)*5)]+
                board[(1)*x+((x/6)*5)+1];

     nextboard[0*x+((x/6)*5)] = cpuDeadorAlive(value,
                                                board[0*x+((x/6)*5)]);

    //bot with front and right
    x =     board[y*x+ (x/6-1)-1]+
            board[y*x+ (x/6-1)]+

            board[0*x+(x - 1)-1]+
            board[0*x+(x - 1)]+
            board[y*x+(x/6)]+

            board[(1)*x+(x - 1)-1]+
            board[(1)*x+(x - 1)]+
            board[(y-1)*x+ (x/6)];
    nextboard[0*x+(x - 1)] = cpuDeadorAlive(value,
                                              board[0*x+(x - 1)]);



    //bot with right and back
    x =     board[y*x+(2*(x/6))]+
            board[y*x+(2*(x/6))+1]+

            board[y*x+(2*(x/6)-1)]+
            board[(y)*x+(x - 1)]+
            board[(y)*x+(x - 1)-1]+

            board[y*x+(2*(x/6)-1)-1]+
            board[(y-1)*x+(x - 1)]+
            board[(y-1)*x+(x - 1)-1];

        nextboard[(y)*x+(x - 1)] = cpuDeadorAlive(value,
                                              board[(y)*x+(x - 1)]);



    //bot with back and left

    x =     board[y*x+(2*(x/6))]+
            board[y*x+(2*(x/6))]+

            board[y*x+(3*(x/6))]+
            board[y*x+((x/6)*5)]+
            board[y*x+((x/6)*5)-1]+

            board[y*x+(3*(x/6))+1]+
            board[(y-1)*x+((x/6)*5)-1]+
            board[(y-1)*x+((x/6)*5)-1];
    nextboard[y*x+((x/6)*5)] = cpuDeadorAlive(value,
                                              board[y*x+((x/6)*5)]);


}

__device__ int get_x_lined(int * board, int col,int x,int y)
{
    return  (board[(y * col) + x-1])+
            (board[(y * col) + x])+
            (board[(y * col) + x+1]);
}

__device__ int cudaAddUpLife(int * board,int col, int x, int y)
{
    return  get_x_lined(board, col, x,y+1)+
            get_x_lined(board, col, x,y)+
            get_x_lined(board, col, x,y-1);
}
__device__ int deadorAlive(int value, int currentLocation)
{
    if (currentLocation == 1)
        if (value - 1 == 2 || value - 1 == 3)
            return 1;
        else
            return 0;
    else if (value == 3)
        return 1;
    else
        return 0;
}


__global__ void two_d_conway_block(int *board, int *next_board, int cols, int x_size, int y_size)
{
    // global thread(data) row index
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // global thread(data) column index
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    int index = y*cols+x;
    int total = 0;

    if((y < y_size) && (x < x_size)) {

        total = cudaAddUpLife(board,cols,x,y);
//        next_board[index] = deadorAlive(total,board[index]);

//        total += board[index-1-cols] + board[index-cols] + board[index + 1 - cols] +
//                 board[index - 1] + board[index] + board[index+1] +
//                 board[index + cols - 1] + board[index + cols] + board[index + cols +1];
        next_board[index] = (total == 3) || (total-(board[index]) == 3);
    }

}


void launch_two_d_conway_block(dim3 blocksize)
{
    const dim3 numBlocksMain((cols)/blocksize.x +1 ,rows/blocksize.y + 1,1);
    const dim3 numBlocks(size/blocksize.x + 1,size/blocksize.y + 1,1);
    two_d_conway_block<<<numBlocksMain, blocksize>>>(d_board + 1 + cols, d_nextboard + 1 + cols, cols, (size*4)-2, size-2);
    two_d_conway_block<<<numBlocks, blocksize>>>(d_board + 1 + size*4 + cols, d_nextboard + 1 + size*4 + cols, cols, size-2, size-2);
    two_d_conway_block<<<numBlocks, blocksize>>>(d_board + 1 + size*5 + cols, d_nextboard + 1 + size*5 + cols, cols, size-2, size-2);
}


__global__ void conway_edges(int *self, int *self_next, int *other, int self_idx_step, int other_idx_step, int adjacent_offset, int len) {
    //Determine the index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx>=len) return;

    //Rebase the indexes for this thread
    self = self + self_idx_step*idx;
    other = other + other_idx_step*idx;
    self_next = self_next + self_idx_step*idx;

    //Get the complete sum
    int sum =   *(other - other_idx_step) + (*other) + *(other + other_idx_step) +
                *(self  - self_idx_step ) + (*self ) + *(self  + self_idx_step ) +
                *(self + adjacent_offset - self_idx_step) + *(self + adjacent_offset) + *(self + adjacent_offset + self_idx_step);

    //Write the result
    *self_next = (sum == 3) || (sum-(*self) == 3);
}

void cuda_launch_conway_edges(int blocksize) {
    const int edge_length = size - 2;
    const dim3 numBlocks(edge_length/blocksize + (1&&(edge_length%blocksize)), 1, 1);

    conway_edges<<<numBlocks, blocksize>>>(d_board + cols, d_nextboard + cols, d_board + size*4 - 1 + cols, cols, cols, 1, edge_length); // front to left
    conway_edges<<<numBlocks, blocksize>>>(d_board + size*4 - 1 + cols, d_nextboard + size*4 - 1 + cols, d_board + cols, cols, cols, -1, edge_length); //left to front

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1, d_nextboard + 1, d_board + (4 * size)+1, 1, 1, cols, edge_length); //top to front
    conway_edges <<<numBlocks, blocksize>>>(d_board + (4 * size)+1, d_nextboard + (4 * size)+1, d_board + 1, 1, 1, cols, edge_length); //front to top

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size*2, d_nextboard + 1 + size*2, d_board + 5*size - 2 + cols*(rows-1), 1, -1, cols, edge_length); //top to back
    conway_edges <<<numBlocks, blocksize>>>(d_board + 5*size - 2 + cols*(rows-1), d_nextboard + 5*size - 2 + cols*(rows-1), d_board + 1 + size*2, -1, 1, -cols, edge_length); //back to top

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size, d_nextboard + 1 + size, d_board + 5*size - 1 + cols, 1, cols, cols, edge_length); //top to right
    conway_edges <<<numBlocks, blocksize>>>(d_board + 5*size - 1 + cols, d_nextboard + 5*size - 1 + cols, d_board + 1 + size, cols, 1, -1, edge_length); //right to top

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size*3, d_nextboard + 1 + size*3, d_board + 4*size + cols*(rows-2), 1, (-cols), cols, edge_length); //top to left
    conway_edges <<<numBlocks, blocksize>>>(d_board + 4*size + cols*(rows-2), d_nextboard + 4*size + cols*(rows-2), d_board + 1 + size*3, (-cols), 1, 1, edge_length); //left to top

    conway_edges <<<numBlocks, blocksize>>>(d_board+cols*(rows-1)+1, d_nextboard+cols*(rows-1)+1, d_board + size*5 + 1, 1, 1, (-cols), edge_length); // bottom to front
    conway_edges <<<numBlocks, blocksize>>>(d_board + size*5 + 1, d_nextboard + size*5 + 1, d_board+cols*(rows-1)+1, 1, 1, (-cols), edge_length); // front to bottom

    conway_edges <<<numBlocks, blocksize>>>(d_board + 1 + size*2, d_nextboard + 1 + size*2, d_board + size*6 - 2 + cols*rows-1, 1, -1, (-cols), edge_length); // bottom to back
    conway_edges <<<numBlocks, blocksize>>>(d_board + size*6 - 2 + cols*rows-1, d_nextboard + size*6 - 2 + cols*rows-1, d_board + 1 + size*2, -1, 1, (-cols), edge_length); // back to bottom
}

// Called when setting things up before graphs loop
__host__ void cudaMainInitialize(int size_set) {
    SPDLOG_INFO("Initialize Cuda");
    float timeStart = 0;
    size = size_set;
    rows = size;
    cols = size*6;

    h_board = (int *) calloc(rows*cols, sizeof(int));
    h_next_board = (int *) calloc(rows*cols, sizeof(int));

    //Initialize a pattern in the conway grid
    /*for (int i = 0; i < cols; ++i) {
        h_board[(3 * cols) + i] = 1;
    }*/

    h_board[3*cols + 10] = 1;
    h_board[3*cols + 11] = 1;
    h_board[3*cols + 12] = 1;
    h_board[4*cols + 10] = 1;
    h_board[5*cols + 11] = 1;

    h_board[3*cols + 10 + size] = 1;
    h_board[3*cols + 11 + size] = 1;
    h_board[3*cols + 12 + size] = 1;
    h_board[4*cols + 10 + size] = 1;
    h_board[5*cols + 11 + size] = 1;

    //cuda memory allocation
    hipMalloc(&d_board, sizeof(int)*rows*cols);
    hipMalloc(&d_nextboard, sizeof(int)*rows*cols);

    hipMemcpy(d_board, h_board, sizeof(int)*rows*cols, hipMemcpyHostToDevice);
}

// Called for every frame
int *cudaMainUpdate() {
    hipError_t code = hipDeviceSynchronize();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Device Synchronize error -- {}", hipGetErrorString(code)));
    }
    // check if kernel execution generated an error
    code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }
    const int blocksize = 32;
    const dim3 blockdim(blocksize, blocksize, 1);

    memcpy(h_next_board, h_board, rows*cols*sizeof(*h_board));

    // #### Corners ####
    //cornerHost(h_board, h_next_board, rows,cols);

    hipMemcpy(d_nextboard, h_next_board, rows*cols*sizeof(*d_nextboard), hipMemcpyHostToDevice);
    hipMemcpy(d_board, h_board, rows*cols*sizeof(*d_nextboard), hipMemcpyHostToDevice);
    // #### Field ####
    launch_two_d_conway_block(blockdim);

    // #### Edges ####
    const int edge_length = size - 2;
    const dim3 numBlocks(edge_length/blocksize + (1&&(edge_length%blocksize)), 1, 1);

    cuda_launch_conway_edges(blocksize);

    hipMemcpy(h_board, d_nextboard, rows*cols*sizeof(*d_nextboard), hipMemcpyDeviceToHost);
    {
        int * temp = d_board;
        d_board = d_nextboard;
        d_nextboard = temp;
    }
    return h_board;
}

// Called as the program close
__host__ void cudaMainCleanUp() {
    SPDLOG_INFO("Clean Up Cuda");

    // check if kernel execution generated an error
    hipError_t code = hipGetLastError();
    if (code != hipSuccess){
        SPDLOG_ERROR(spdlog::fmt_lib::format("Cuda Kernel Launch error -- {}", hipGetErrorString(code)));
    }

    hipFree(d_board);
    hipFree(d_nextboard);
    free(h_board);
    free(h_next_board);
}